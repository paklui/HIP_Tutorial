#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// GPU  kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
    //Host output vector for verification
    double *h_verify_c;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
    h_verify_c = (double*)malloc(bytes);
    
    // Allocate memory for each vector on GPU
   hipMalloc(&d_a, bytes);
   hipMalloc(&d_b, bytes);
   hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = i;
        h_b[i] = i;
    }


 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes,  hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    hipLaunchKernelGGL(vecAdd, dim3(gridSize), dim3(blockSize), 0, 0, d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    // Copy array back to host
   hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost);
 

   //Compute for CPU 
   for(i=0; i <n; i++)
   {
    h_verify_c[i] = h_a[i] + h_b[i];
   }


    //Verfiy results
    for(i=0; i <n; i++)
    {
    if (abs(h_verify_c[i] - h_c[i]) > 1e-5) 
     {
     printf("Error at position i %d, Expected: %f, Found: %f \n", i, h_c[i], d_c[i]);
     }  
    }


     printf("Printing a subset of results till index 1024\n");

     for(i = 0; i < 1024 ; i++)
     {
	printf("Value at index %d is %f\n",i, h_c[i]);
     }

    
     
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
