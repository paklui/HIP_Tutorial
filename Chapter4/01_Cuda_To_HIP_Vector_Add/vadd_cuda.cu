
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// GPU  kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
    //Host output vector for verification
    double *h_verify_c;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
    h_verify_c = (double*)malloc(bytes);
    
    // Allocate memory for each vector on GPU
   hipMalloc(&d_a, bytes);
   hipMalloc(&d_b, bytes);
   hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = i;
        h_b[i] = i;
    }


 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes,  hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    // Copy array back to host
   hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost);
 

   //Compute for CPU 
   for(i=0; i <n; i++)
   {
    h_verify_c[i] = h_a[i] + h_b[i];
   }


    //Verfiy results
    for(i=0; i <n; i++)
    {
    if (abs(h_verify_c[i] - h_c[i]) > 1e-5) 
     {
     printf("Error at position i %d, Expected: %f, Found: %f \n", i, h_c[i], d_c[i]);
     }  
    }


     printf("Printing a subset of results till index 1024\n");

     for(i = 0; i < 1024 ; i++)
     {
	printf("Value at index %d is %f\n",i, h_c[i]);
     }

    
     
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
